#include "hip/hip_runtime.h"
//Math Libraries
#include <complex>
#include <cmath>
#include <cfloat>

//CUDA libraries
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

//OpenMP
#include <omp.h>

//IPT Solver
#include "SIAM.h"
#include "Grid.h"
#include "dinterpl.h"

//GSL Libraries for Adaptive Cauchy Integration
#include <gsl/gsl_errno.h>
#include <gsl/gsl_integration.h>

// Device constants header //
//__constant__ <type> <variable>;

// Constants
const int MAX_GPU_COUNT=8;
const int CUDA_threadcount = 250;

// Typedef

// Structs

struct KK_params
    {
      double* omega;
      double* Y;
      int N;
      dinterpl* spline;
    };

template<typename T> struct TGPUplan
{
    //Host-side input data
    int N_LOCAL,i0;
    int block_count;
    
    //Host buffers
    T *h_P1,*h_P2,*h_imSOCSigma;

    //Device buffers
    T *d_omega,*d_Ap,*d_Am,*d_P1,*d_P2,*d_imSOCSigma;

    //Stream for asynchronous command execution
    hipStream_t stream;

} ;

/* GPU Function Headers */

// Polarization Functions
template<typename T> __global__ void d_get_Ps(T P1[], T P2[], const T Ap[], const T Am[], const T omega[], int i0, int N_LOCAL, int N);

// Imaginary Part of Second-order correction to self energy
template<typename T> __global__ void d_get_imSOCSigma(T imSOCSigma[],const T P1[], const T P2[], const T Ap[] , const T Am[] , const T omega[], T U, int i0, int N_LOCAL, int N);

// Interpolation Function
template<typename T> __device__ int d_interp_bsearch (const T x_array[], T x, int index_lo, int index_hi);
template<typename T> __device__ T d_interpl(T x, const T y_array[],  const T x_array[], int size);

/* CPU Function Headers */

//Interpolation
size_t interp_bsearch (const double x_array[], double x, int index_lo, int index_hi);
double linear_interpl(double x, const double y_array[],  const double x_array[], int size);

//Clipoff
bool h_ClipOff(double &X);
double imSOCSigmafc(double om, void *params);
double imSOCSigmafl(double om, void *params);


//Interpolation Function
    
inline size_t interp_bsearch (const double x_array[], double x, int index_lo, int index_hi) 
    {
      size_t min = index_lo, max = index_hi;
      while (min + 1 < max)
      {
        size_t i = (min + max) >> 1;
        min = x > x_array[i] ? i : min;
        max = x > x_array[i] ? max : i;
      }
      return min;
    }
    
double linear_interpl (double x, const double x_array[], const double y_array[], int size){

  double x_lo, x_hi;
  double y_lo, y_hi;
  double y;
  size_t index;
  const double xmin = x_array[0];
  const double xmax = x_array[size-1];

  if (x > xmax or x < xmin) {
    y = 0.0;
  }
  else {
    index = interp_bsearch (x_array, x, 0, size - 1);

    /* evaluate */
    x_lo = x_array[index];
    x_hi = x_array[index + 1];
    y_lo = y_array[index];
    y_hi = y_array[index + 1];

    y = y_lo + (x - x_lo) / (x_hi - x_lo) * (y_hi - y_lo);
  }
  return y;
}

double imSOCSigmafc(double om, void *params)
{
  struct KK_params *p= (struct KK_params *)params;
  return p->spline->cspline_eval(om);
}

double imSOCSigmafl(double om, void *params)
{
  struct KK_params *p= (struct KK_params *)params;
  return dinterpl::linear_eval(om,p->omega, p->Y , p->N);
}

bool h_ClipOff(double &X)
{
  if (X>0) 
  {
    X = -CLIPVAL;
    return true;
  }
  else
    return false;
}

/*=================================== DEVICE INTERPOLATION FUNCTIONS ===================================*/

template<typename T> __device__ inline int d_interp_bsearch (const T x_array[], T x, int index_lo, int index_hi)
{
  int min = index_lo, max = index_hi;
  while (min + 1 < max)
  {
    int i = (min + max) >> 1;
    min = x > x_array[i] ? i : min;
    max = x > x_array[i] ? max : i;
  }
  return min;
}

template<typename T> __device__ T d_interpl(T x, const T y_array[],  const T x_array[], int size)
{
  T x_lo, x_hi;
  T y_lo, y_hi;
  T y;
  int index;
  const T xmin = x_array[0];
  const T xmax = x_array[size-1];

  if (x > xmax or x < xmin) {
    y = 0.0;
  }
  else {
    index = d_interp_bsearch<T>(x_array, x, 0, size - 1);

    /* evaluate */
    x_lo = x_array[index];
    x_hi = x_array[index + 1];
    y_lo = y_array[index];
    y_hi = y_array[index + 1];

    y = y_lo + (x - x_lo) / (x_hi - x_lo) * (y_hi - y_lo);
  }
  return y;
}


/*=================================== DEVICE SIAM FUNCTIONS ===================================*/

template<typename T> __global__ void d_get_Ps(T P1[], T P2[], const T Ap[], const T Am[], const T omega[], int i0, int N_LOCAL, int N)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int i = i0+tid;

  if (tid < N_LOCAL){

		//Trapezoidal Integral (range from 0 to N-1)
		
		T p1sum = 0.0;
		T p2sum = 0.0;
		
		const T p1_0 = Am[0]* d_interpl<T>(omega[0] - omega[i], Ap, omega,N);
		const T p2_0 = Ap[0]* d_interpl<T>(omega[0] - omega[i], Am, omega,N);
		const T p1_N = Am[N]* d_interpl<T>(omega[N-1] - omega[i], Ap, omega,N);
		const T p2_N = Ap[N]* d_interpl<T>(omega[N-1] - omega[i], Am, omega,N);
		
		p1sum += p1_0*(omega[1]-omega[0])+p1_N*(omega[N-1]-omega[N-2]);
		p1sum += p2_0*(omega[1]-omega[0])+p2_N*(omega[N-1]-omega[N-2]);

		for (int j=1;j<N-1;j++){
			const T p1 = Am[j]* d_interpl<T>(omega[j] - omega[i], Ap, omega,N);
			const T p2 = Ap[j]* d_interpl<T>(omega[j] - omega[i], Am, omega,N);
			const T domega = (omega[j+1]-omega[j-1]);
			p1sum += domega*p1;
			p2sum += domega*p2;
		}

		//Store result (to be pieced together if multi-GPU)
		P1[tid] = M_PI * p1sum * 0.5;
		P2[tid] = M_PI * p2sum * 0.5;
		
  }
}

template<typename T> __global__ void d_get_imSOCSigma(T imSOCSigma[],const T P1[], const T P2[], const T Ap[] , const T Am[] , const T omega[], T U, int i0, int N_LOCAL, int N)
{

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int i = i0+tid;

  if (tid < N_LOCAL)
  {
		//Trapezoidal Integral (range from 0 to N-1)
  	T ssum = 0.0;
  	
    const T s_0 = d_interpl<T>(omega[i] - omega[0], Ap, omega,N) * P2[0] + d_interpl<T>(omega[i] - omega[0], Am, omega,N) * P1[0];
    const T s_N = d_interpl<T>(omega[i] - omega[N-1], Ap, omega,N) * P2[N-1] + d_interpl<T>(omega[i] - omega[N-1], Am, omega,N) * P1[N-1];
  	
  	ssum += s_0*(omega[1]-omega[0])+s_N*(omega[N-1]-omega[N-2]);
  	
  	
    for (int j=1; j<N-1; j++) 
    { 
				const T s = d_interpl<T>(omega[i] - omega[j], Ap, omega,N) * P2[j] + d_interpl<T>(omega[i] - omega[j], Am, omega,N) * P1[j];
				ssum += (omega[j+1]-omega[j-1])*s;
    }            
      
    imSOCSigma[tid] = - U*U * ssum * 0.5;
  }
}

//=================================== HOST SIAM FUNCTIONS (FP64) ===================================//

void SIAM::get_Ps()
{

  TGPUplan<double> plan[MAX_GPU_COUNT];
    
  //Initialize multi-GPUs
  int GPU_N;
  hipGetDeviceCount(&GPU_N); //Get amount of GPUs, assuming to be > 0;

  if (GPU_N > MAX_GPU_COUNT)
  {
    	GPU_N = MAX_GPU_COUNT;
  }
  
	//Subdividing input data across GPUs
	//Get data sizes for each GPU
	for (int i = 0; i < GPU_N; i++)
	{
			plan[i].N_LOCAL = N / GPU_N;
	}
	
	//Take into account "odd" data sizes
	for (int i = 0; i < N % GPU_N; i++)
	{
			plan[i].N_LOCAL++;
	}
	
	//Begining omega-point for each GPU
	int cumN=0;
	for (int i = 0; i<GPU_N ;i++)
	{
			plan[i].i0 = cumN;
			cumN += plan[i].N_LOCAL;
	}
	
	for (int i = 0; i<GPU_N ;i++)
	{
	    hipSetDevice(i);
	    hipStreamCreate(&plan[i].stream);
	    
    	//Allocate memory
	    hipMalloc((void **)&plan[i].d_omega, N * sizeof(double));
	    hipMalloc((void **)&plan[i].d_Ap, N * sizeof(double));
	    hipMalloc((void **)&plan[i].d_Am, N * sizeof(double));
	    hipMalloc((void **)&plan[i].d_P1, plan[i].N_LOCAL * sizeof(double));
	    hipMalloc((void **)&plan[i].d_P2, plan[i].N_LOCAL * sizeof(double));
	    hipHostMalloc((void **)&plan[i].h_P1, plan[i].N_LOCAL * sizeof(double));
	    hipHostMalloc((void **)&plan[i].h_P2, plan[i].N_LOCAL * sizeof(double));
	    
			//initialize block count
			plan[i].block_count = (plan[i].N_LOCAL + CUDA_threadcount) / CUDA_threadcount;
	}
	
  for (int i = 0; i < GPU_N; i++)
  {
  		//Create streams for issuing GPU command asynchronously
	    hipSetDevice(i);
	   
			
			//Copy omega-grid from CPU
			hipMemcpyAsync(plan[i].d_omega, g->omega, N * sizeof(double), hipMemcpyHostToDevice, plan[i].stream);
      
			//Copy Ap,Am from CPU
			hipMemcpyAsync(plan[i].d_Ap, g->Ap, N * sizeof(double), hipMemcpyHostToDevice, plan[i].stream);
			hipMemcpyAsync(plan[i].d_Am, g->Am, N * sizeof(double), hipMemcpyHostToDevice, plan[i].stream);
			
      //Perform GPU computations
  		d_get_Ps<double><<<plan[i].block_count,CUDA_threadcount,0,plan[i].stream>>>(plan[i].d_P1, plan[i].d_P2, plan[i].d_Ap, plan[i].d_Am, plan[i].d_omega,plan[i].i0,plan[i].N_LOCAL, N);

      //Read back GPU results
      hipMemcpyAsync(plan[i].h_P1, plan[i].d_P1, plan[i].N_LOCAL *sizeof(double), hipMemcpyDeviceToHost, plan[i].stream);
      hipMemcpyAsync(plan[i].h_P2, plan[i].d_P2, plan[i].N_LOCAL *sizeof(double), hipMemcpyDeviceToHost, plan[i].stream);
  }
  
  //Process GPU results
  for (int i = 0; i < GPU_N; i++)
  {
      //Set device
      hipSetDevice(i);

      //Wait for all operations to finish
      hipStreamSynchronize(plan[i].stream);

			//printf("Finished...\n");
      //Piece together results from GPU

      for (int j = 0; j < plan[i].N_LOCAL; j++)
      {
          g->P1[plan[i].i0+j] = plan[i].h_P1[j];
          g->P2[plan[i].i0+j] = plan[i].h_P2[j];
      }
      
			//printf("Freeing memory...\n");
			
      hipHostFree(plan[i].h_P1);
      hipHostFree(plan[i].h_P2);
      hipFree(plan[i].d_omega);
      hipFree(plan[i].d_Ap);
      hipFree(plan[i].d_Am);
      hipFree(plan[i].d_P1);
      hipFree(plan[i].d_P2);
      hipStreamDestroy(plan[i].stream);
  }
  
  hipDeviceSynchronize();
  
}


void SIAM::get_SOCSigma(){

  double *imSOCSigma = new double[N];
  
  TGPUplan<double> plan[MAX_GPU_COUNT];
    
  //Initialize multi-GPUs
  int GPU_N;
  hipGetDeviceCount(&GPU_N); //Get amount of GPUs, assuming to be > 0;

  if (GPU_N > MAX_GPU_COUNT)
  {
    	GPU_N = MAX_GPU_COUNT;
  }
  
	//Subdividing input data across GPUs
	//Get data sizes for each GPU
	for (int i = 0; i < GPU_N; i++)
	{
			plan[i].N_LOCAL = N / GPU_N;
	}
	
	//Take into account "odd" data sizes
	for (int i = 0; i < N % GPU_N; i++)
	{
			plan[i].N_LOCAL++;
	}
	
	//Begining omega-point for each GPU
	int cumN=0;
	for (int i = 0; i<GPU_N ;i++)
	{
			plan[i].i0 = cumN;
			cumN += plan[i].N_LOCAL;
			//printf("GPU %d will work on %d points from %d\n",i,plan[i].N_LOCAL,plan[i].i0);
	}

  //allocate memory (GPU and System page-locked)
  for (int i = 0; i < GPU_N; i++)
  {
      hipSetDevice(i);
      
	    hipStreamCreate(&plan[i].stream);
      
      //Allocate memory
	    hipMalloc((void **)&plan[i].d_omega, N * sizeof(double));
	    hipMalloc((void **)&plan[i].d_Ap, N * sizeof(double));
	    hipMalloc((void **)&plan[i].d_Am, N * sizeof(double));
      hipMalloc((void **)&plan[i].d_P1, N * sizeof(double));
      hipMalloc((void **)&plan[i].d_P2, N * sizeof(double));
      hipMalloc((void **)&plan[i].d_imSOCSigma, plan[i].N_LOCAL * sizeof(double));
      hipHostMalloc((void **)&plan[i].h_imSOCSigma, plan[i].N_LOCAL * sizeof(double));
      
			//initialize block count
			plan[i].block_count = (plan[i].N_LOCAL + CUDA_threadcount) / CUDA_threadcount;
  }
  
  //Copy data to GPU, launch the kernel and copy data back. All asynchronously
  for (int i = 0; i < GPU_N; i++)
  {
      //Set device
      hipSetDevice(i);
      
      
      //Copy pieced-together P1,P2 from CPU
      hipMemcpyAsync(plan[i].d_omega, g->omega, N * sizeof(double), hipMemcpyHostToDevice, plan[i].stream);
      hipMemcpyAsync(plan[i].d_Ap, g->Ap, N * sizeof(double), hipMemcpyHostToDevice, plan[i].stream);
      hipMemcpyAsync(plan[i].d_Am, g->Am, N * sizeof(double), hipMemcpyHostToDevice, plan[i].stream);
      hipMemcpyAsync(plan[i].d_P1, g->P1, N * sizeof(double), hipMemcpyHostToDevice, plan[i].stream);
      hipMemcpyAsync(plan[i].d_P2, g->P2, N * sizeof(double), hipMemcpyHostToDevice, plan[i].stream);

      //Perform GPU computations
  		d_get_imSOCSigma<double><<<plan[i].block_count,CUDA_threadcount,0,plan[i].stream>>>(plan[i].d_imSOCSigma, plan[i].d_P1, plan[i].d_P2, plan[i].d_Ap , plan[i].d_Am , plan[i].d_omega, U,
  																																								plan[i].i0, plan[i].N_LOCAL, N);

      //Read back GPU results
      hipMemcpyAsync(plan[i].h_imSOCSigma, plan[i].d_imSOCSigma, plan[i].N_LOCAL * sizeof(double), hipMemcpyDeviceToHost, plan[i].stream);

  }
  
  //Process GPU results
  for (int i = 0; i < GPU_N; i++)
  {
      //Set device
      hipSetDevice(i);

      //Wait for all operations to finish
      hipStreamSynchronize(plan[i].stream);

      //Piece together results from GPU

      for (int j = 0; j < plan[i].N_LOCAL; j++)
      {
          imSOCSigma[plan[i].i0+j] = plan[i].h_imSOCSigma[j];
          if (h_ClipOff( imSOCSigma[plan[i].i0+j] )) Clipped = true ;
      }
      
      hipHostFree(plan[i].h_imSOCSigma);
      hipFree(plan[i].d_omega);
      hipFree(plan[i].d_Ap);
      hipFree(plan[i].d_Am);
      hipFree(plan[i].d_P1);
      hipFree(plan[i].d_P2);
      hipFree(plan[i].d_imSOCSigma);
      hipStreamDestroy(plan[i].stream);
  }
  if (Clipped) sprintf(ibuffer + strlen(ibuffer),"SIAM::run::(Warning) !!!Clipping SOCSigma!!!!\n");
  
  hipDeviceSynchronize();  
  
  // The KramersKonig function is not used to compute the Cauchy Integral, this allows arbitrary grid to be used.
  
  gsl_set_error_handler_off();
  #pragma omp parallel for schedule(dynamic)
  for (int i=1; i<N-1; i++)
  { 
    const double a = g->omega[0], b = g->omega[N-1]; // limits of integration
    const double epsabs = 0, epsrel = KKAccr; // requested errors
    double result; // the integral value
    double error; // the error estimate

    double c = g->omega[i];

    struct KK_params params;
    gsl_function F;
    
		dinterpl spline(g->omega, imSOCSigma , N);
    if (!usecubicspline){
		  params.omega = g->omega;
		  params.Y = imSOCSigma;
		  params.N = N;
    	F.function = &imSOCSigmafl;
    }
    else{
    	params.spline = &spline;
    	F.function = &imSOCSigmafc;
    }

    //F.function = &imSOCSigmafc;
    F.params = &params;

    
    size_t limit = QUADLIMIT;// work area size
    gsl_integration_workspace *ws = gsl_integration_workspace_alloc (limit);

    gsl_integration_qawc (&F, a, b , c , epsabs, epsrel, limit, ws, &result, &error);

    gsl_integration_workspace_free (ws);

    g->SOCSigma[i] = complex<double>(result/M_PI,imSOCSigma[i]);
  }
  
  g->SOCSigma[0] = g->SOCSigma[1];
  g->SOCSigma[N-1] = g->SOCSigma[N-2];

  delete [] imSOCSigma;
}


//=================================== HOST SIAM FUNCTIONS (FP32, use if FP64 is too slow on consumer grade GPUs) ===================================//
/*
void SIAM::get_Ps()
{

  TGPUplan<float> plan[MAX_GPU_COUNT];
  
  //Type cast double to float
  float *omega32 = new float[N];
  float *Ap32 = new float[N];
  float *Am32 = new float[N];
  std::copy(g->omega,(g->omega)+N,omega32);
  std::copy(g->Ap,(g->Ap)+N,Ap32);
  std::copy(g->Am,(g->Am)+N,Am32);
    
  //Initialize multi-GPUs
  int GPU_N;
  hipGetDeviceCount(&GPU_N); //Get amount of GPUs, assuming to be > 0;

  if (GPU_N > MAX_GPU_COUNT)
  {
    	GPU_N = MAX_GPU_COUNT;
  }
  
	//Subdividing input data across GPUs
	//Get data sizes for each GPU
	for (int i = 0; i < GPU_N; i++)
	{
			plan[i].N_LOCAL = N / GPU_N;
	}
	
	//Take into account "odd" data sizes
	for (int i = 0; i < N % GPU_N; i++)
	{
			plan[i].N_LOCAL++;
	}
	
	//Begining omega-point for each GPU
	int cumN=0;
	for (int i = 0; i<GPU_N ;i++)
	{
			plan[i].i0 = cumN;
			cumN += plan[i].N_LOCAL;
			//printf("GPU %d will work on %d points from %d\n",i,plan[i].N_LOCAL,plan[i].i0);
	}
	
	for (int i = 0; i<GPU_N ;i++)
	{
	    hipSetDevice(i);
	    hipStreamCreate(&plan[i].stream);
	    
    	//Allocate memory
	    hipMalloc((void **)&plan[i].d_omega, N * sizeof(float));
	    hipMalloc((void **)&plan[i].d_Ap, N * sizeof(float));
	    hipMalloc((void **)&plan[i].d_Am, N * sizeof(float));
	    hipMalloc((void **)&plan[i].d_P1, plan[i].N_LOCAL * sizeof(float));
	    hipMalloc((void **)&plan[i].d_P2, plan[i].N_LOCAL * sizeof(float));
	    hipHostMalloc((void **)&plan[i].h_P1, plan[i].N_LOCAL * sizeof(float));
	    hipHostMalloc((void **)&plan[i].h_P2, plan[i].N_LOCAL * sizeof(float));
	    
			//initialize block count
			plan[i].block_count = (plan[i].N_LOCAL + CUDA_threadcount) / CUDA_threadcount;
	}
	
  for (int i = 0; i < GPU_N; i++)
  {
  		//Create streams for issuing GPU command asynchronously
	    hipSetDevice(i);
	   
			
			//Copy omega-grid from CPU
			hipMemcpyAsync(plan[i].d_omega, omega32, N * sizeof(float), hipMemcpyHostToDevice, plan[i].stream);
      
			//Copy Ap,Am from CPU
			hipMemcpyAsync(plan[i].d_Ap, Ap32, N * sizeof(float), hipMemcpyHostToDevice, plan[i].stream);
			hipMemcpyAsync(plan[i].d_Am, Am32, N * sizeof(float), hipMemcpyHostToDevice, plan[i].stream);
			
      //Perform GPU computations
  		d_get_Ps<float><<<plan[i].block_count,CUDA_threadcount,0,plan[i].stream>>>(plan[i].d_P1, plan[i].d_P2, plan[i].d_Ap, plan[i].d_Am, plan[i].d_omega,plan[i].i0,plan[i].N_LOCAL, N);

      //Read back GPU results
      hipMemcpyAsync(plan[i].h_P1, plan[i].d_P1, plan[i].N_LOCAL *sizeof(float), hipMemcpyDeviceToHost, plan[i].stream);
      hipMemcpyAsync(plan[i].h_P2, plan[i].d_P2, plan[i].N_LOCAL *sizeof(float), hipMemcpyDeviceToHost, plan[i].stream);
  }
  
  //Process GPU results
  for (int i = 0; i < GPU_N; i++)
  {
      //Set device
      hipSetDevice(i);

      //Wait for all operations to finish
      hipStreamSynchronize(plan[i].stream);

      //Piece together results from GPU (typecast to double)
      for (int j = 0; j < plan[i].N_LOCAL; j++)
      {
          g->P1[plan[i].i0+j] = (double) plan[i].h_P1[j];
          g->P2[plan[i].i0+j] = (double) plan[i].h_P2[j];
      }
      
      hipHostFree(plan[i].h_P1);
      hipHostFree(plan[i].h_P2);
      hipFree(plan[i].d_omega);
      hipFree(plan[i].d_Ap);
      hipFree(plan[i].d_Am);
      hipFree(plan[i].d_P1);
      hipFree(plan[i].d_P2);
      hipStreamDestroy(plan[i].stream);
  }
  hipDeviceSynchronize();
  delete [] omega32;
  delete [] Ap32;
  delete [] Am32;
}


void SIAM::get_SOCSigma(){

  double *imSOCSigma = new double[N];
  
  TGPUplan<float> plan[MAX_GPU_COUNT];
  
  //Type cast double to float
  float *omega32 = new float[N];
  float *Ap32 = new float[N];
  float *Am32 = new float[N];
  float *P132 = new float[N];
  float *P232 = new float[N];
  std::copy(g->omega,(g->omega)+N,omega32);
  std::copy(g->Ap,(g->Ap)+N,Ap32);
  std::copy(g->Am,(g->Am)+N,Am32);
  std::copy(g->P1,(g->P1)+N,P132);
  std::copy(g->P2,(g->P2)+N,P232);
    
  //Initialize multi-GPUs
  int GPU_N;
  hipGetDeviceCount(&GPU_N); //Get amount of GPUs, assuming to be > 0;

  if (GPU_N > MAX_GPU_COUNT)
  {
    	GPU_N = MAX_GPU_COUNT;
  }
  
	//Subdividing input data across GPUs
	//Get data sizes for each GPU
	for (int i = 0; i < GPU_N; i++)
	{
			plan[i].N_LOCAL = N / GPU_N;
	}
	
	//Take into account "odd" data sizes
	for (int i = 0; i < N % GPU_N; i++)
	{
			plan[i].N_LOCAL++;
	}
	
	//Begining omega-point for each GPU
	int cumN=0;
	for (int i = 0; i<GPU_N ;i++)
	{
			plan[i].i0 = cumN;
			cumN += plan[i].N_LOCAL;
			//printf("GPU %d will work on %d points from %d\n",i,plan[i].N_LOCAL,plan[i].i0);
	}

  //allocate memory (GPU and System page-locked)
  for (int i = 0; i < GPU_N; i++)
  {
      hipSetDevice(i);
      
	    hipStreamCreate(&plan[i].stream);
      
      //Allocate memory
	    hipMalloc((void **)&plan[i].d_omega, N * sizeof(float));
	    hipMalloc((void **)&plan[i].d_Ap, N * sizeof(float));
	    hipMalloc((void **)&plan[i].d_Am, N * sizeof(float));
      hipMalloc((void **)&plan[i].d_P1, N * sizeof(float));
      hipMalloc((void **)&plan[i].d_P2, N * sizeof(float));
      hipMalloc((void **)&plan[i].d_imSOCSigma, plan[i].N_LOCAL * sizeof(float));
      hipHostMalloc((void **)&plan[i].h_imSOCSigma, plan[i].N_LOCAL * sizeof(float));
      
			//initialize block count
			plan[i].block_count = (plan[i].N_LOCAL + CUDA_threadcount) / CUDA_threadcount;
  }
  
  //Copy data to GPU, launch the kernel and copy data back. All asynchronously
  for (int i = 0; i < GPU_N; i++)
  {
      //Set device
      hipSetDevice(i);
      
      
      //Copy pieced-together P1,P2 from CPU
      hipMemcpyAsync(plan[i].d_omega, omega32, N * sizeof(float), hipMemcpyHostToDevice, plan[i].stream);
      hipMemcpyAsync(plan[i].d_Ap, Ap32, N * sizeof(float), hipMemcpyHostToDevice, plan[i].stream);
      hipMemcpyAsync(plan[i].d_Am, Am32, N * sizeof(float), hipMemcpyHostToDevice, plan[i].stream);
      hipMemcpyAsync(plan[i].d_P1, P132, N * sizeof(float), hipMemcpyHostToDevice, plan[i].stream);
      hipMemcpyAsync(plan[i].d_P2, P232, N * sizeof(float), hipMemcpyHostToDevice, plan[i].stream);

      //Perform GPU computations
  		d_get_imSOCSigma<float><<<plan[i].block_count,CUDA_threadcount,0,plan[i].stream>>>(plan[i].d_imSOCSigma, plan[i].d_P1, plan[i].d_P2, plan[i].d_Ap , plan[i].d_Am , plan[i].d_omega, U,
  																																								plan[i].i0, plan[i].N_LOCAL, N);

      //Read back GPU results
      hipMemcpyAsync(plan[i].h_imSOCSigma, plan[i].d_imSOCSigma, plan[i].N_LOCAL * sizeof(float), hipMemcpyDeviceToHost, plan[i].stream);

  }
  
  //Process GPU results
  for (int i = 0; i < GPU_N; i++)
  {
      //Set device
      hipSetDevice(i);

      //Wait for all operations to finish
      hipStreamSynchronize(plan[i].stream);

      //Piece together results from GPU (typecast to double)
      for (int j = 0; j < plan[i].N_LOCAL; j++)
      {
          imSOCSigma[plan[i].i0+j] = (double) plan[i].h_imSOCSigma[j];
          if (h_ClipOff( imSOCSigma[plan[i].i0+j] )) Clipped = true ;
      }
      
      hipHostFree(plan[i].h_imSOCSigma);
      hipFree(plan[i].d_omega);
      hipFree(plan[i].d_Ap);
      hipFree(plan[i].d_Am);
      hipFree(plan[i].d_P1);
      hipFree(plan[i].d_P2);
      hipFree(plan[i].d_imSOCSigma);
      hipStreamDestroy(plan[i].stream);
  }
  if (Clipped) printf("    !!!Clipping SOCSigma!!!!\n");
  
  hipDeviceSynchronize();  
  
  // The KramersKonig function is not used to compute the Cauchy Integral, this allows arbitrary grid to be used.
  
  gsl_set_error_handler_off();
  #pragma omp parallel for schedule(dynamic)
  for (int i=1; i<N-1; i++)
  { 
    const double a = g->omega[0], b = g->omega[N-1]; // limits of integration
    const double epsabs = 0., epsrel = 1.e-12; // requested errors
    double result; // the integral value
    double error; // the error estimate

    double c = g->omega[i];

    struct KK_params params;
    params.omega = g->omega;
    params.Y = imSOCSigma;
    params.N = N;

    gsl_function F;
    F.function = &imSOCSigmafc;
    F.params = &params;

    
    size_t limit = 100;// work area size
    gsl_integration_workspace *ws = gsl_integration_workspace_alloc (limit);

    gsl_integration_qawc (&F, a, b , c , epsabs, epsrel, limit, ws, &result, &error);

    gsl_integration_workspace_free (ws);

    g->SOCSigma[i] = complex<double>(result/M_PI,imSOCSigma[i]);
  }
  
  g->SOCSigma[0] = g->SOCSigma[1];
  g->SOCSigma[N-1] = g->SOCSigma[N-2];

  delete [] imSOCSigma;
  
  delete [] omega32;
  delete [] Ap32;
  delete [] Am32;
  delete [] P132;
  delete [] P232;
}

*/
